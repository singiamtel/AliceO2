#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///

#include <hip/hip_runtime.h>
#include <array>
#include <sstream>
#include <iostream>
#include <unistd.h>
#include <thread>

#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/unique.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>

#include "ITStracking/Constants.h"
#include "ITStracking/Configuration.h"
#include "ITStracking/IndexTableUtils.h"
#include "ITStracking/MathUtils.h"
#include "DataFormatsITS/TrackITS.h"
#include "ReconstructionDataFormats/Vertex.h"

#include "ITStrackingGPU/TrackerTraitsGPU.h"
#include "ITStrackingGPU/TrackingKernels.h"

#ifndef __HIPCC__
#define THRUST_NAMESPACE thrust::cuda
#else
#define THRUST_NAMESPACE thrust::hip
#endif

#ifdef GPUCA_NO_FAST_MATH
#define GPU_BLOCKS 1
#define GPU_THREADS 1
#else
#define GPU_BLOCKS 99999
#define GPU_THREADS 99999
#endif

// O2 track model
#include "ReconstructionDataFormats/Track.h"
#include "DetectorsBase/Propagator.h"
using namespace o2::track;

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    LOGF(error, "GPUassert: %s %s %d", hipGetErrorString(code), file, line);
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}

namespace o2::its
{
using namespace constants::its2;
using Vertex = o2::dataformats::Vertex<o2::dataformats::TimeStamp<int>>;

GPUd() float Sq(float v)
{
  return v * v;
}

namespace gpu
{

GPUd() const int4 getBinsRect(const Cluster& currentCluster, const int layerIndex,
                              const o2::its::IndexTableUtils& utils,
                              const float z1, const float z2, float maxdeltaz, float maxdeltaphi)
{
  const float zRangeMin = o2::gpu::CAMath::Min(z1, z2) - maxdeltaz;
  const float phiRangeMin = (maxdeltaphi > constants::math::Pi) ? 0.f : currentCluster.phi - maxdeltaphi;
  const float zRangeMax = o2::gpu::CAMath::Max(z1, z2) + maxdeltaz;
  const float phiRangeMax = (maxdeltaphi > constants::math::Pi) ? constants::math::TwoPi : currentCluster.phi + maxdeltaphi;

  if (zRangeMax < -LayersZCoordinate()[layerIndex + 1] ||
      zRangeMin > LayersZCoordinate()[layerIndex + 1] || zRangeMin > zRangeMax) {

    return getEmptyBinsRect();
  }

  return int4{o2::gpu::CAMath::Max(0, utils.getZBinIndex(layerIndex + 1, zRangeMin)),
              utils.getPhiBinIndex(math_utils::getNormalizedPhi(phiRangeMin)),
              o2::gpu::CAMath::Min(ZBins - 1, utils.getZBinIndex(layerIndex + 1, zRangeMax)),
              utils.getPhiBinIndex(math_utils::getNormalizedPhi(phiRangeMax))};
}

GPUd() bool fitTrack(TrackITSExt& track,
                     int start,
                     int end,
                     int step,
                     float chi2clcut,
                     float chi2ndfcut,
                     float maxQoverPt,
                     int nCl,
                     float bz,
                     const TrackingFrameInfo** tfInfos,
                     const o2::base::Propagator* prop,
                     o2::base::PropagatorF::MatCorrType matCorrType)
{
  for (int iLayer{start}; iLayer != end; iLayer += step) {
    if (track.getClusterIndex(iLayer) == constants::its::UnusedIndex) {
      continue;
    }
    const TrackingFrameInfo& trackingHit = tfInfos[iLayer][track.getClusterIndex(iLayer)];
    if (!track.o2::track::TrackParCovF::rotate(trackingHit.alphaTrackingFrame)) {
      return false;
    }

    if (!prop->propagateToX(track,
                            trackingHit.xTrackingFrame,
                            bz,
                            o2::base::PropagatorImpl<float>::MAX_SIN_PHI,
                            o2::base::PropagatorImpl<float>::MAX_STEP,
                            matCorrType)) {
      return false;
    }

    if (matCorrType == o2::base::PropagatorF::MatCorrType::USEMatCorrNONE) {
      const float xx0 = (iLayer > 2) ? 1.e-2f : 5.e-3f; // Rough layer thickness
      constexpr float radiationLength = 9.36f;          // Radiation length of Si [cm]
      constexpr float density = 2.33f;                  // Density of Si [g/cm^3]
      if (!track.correctForMaterial(xx0, xx0 * radiationLength * density, true)) {
        return false;
      }
    }

    auto predChi2{track.getPredictedChi2(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)};

    if ((nCl >= 3 && predChi2 > chi2clcut) || predChi2 < 0.f) {
      return false;
    }
    track.setChi2(track.getChi2() + predChi2);
    if (!track.o2::track::TrackParCov::update(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)) {
      return false;
    }
    nCl++;
  }
  return o2::gpu::CAMath::Abs(track.getQ2Pt()) < maxQoverPt && track.getChi2() < chi2ndfcut * (nCl * 2 - 5);
}

GPUd() o2::track::TrackParCov buildTrackSeed(const Cluster& cluster1,
                                             const Cluster& cluster2,
                                             const TrackingFrameInfo& tf3,
                                             const float bz)
{
  const float ca = o2::gpu::CAMath::Cos(tf3.alphaTrackingFrame), sa = o2::gpu::CAMath::Sin(tf3.alphaTrackingFrame);
  const float x1 = cluster1.xCoordinate * ca + cluster1.yCoordinate * sa;
  const float y1 = -cluster1.xCoordinate * sa + cluster1.yCoordinate * ca;
  const float z1 = cluster1.zCoordinate;
  const float x2 = cluster2.xCoordinate * ca + cluster2.yCoordinate * sa;
  const float y2 = -cluster2.xCoordinate * sa + cluster2.yCoordinate * ca;
  const float z2 = cluster2.zCoordinate;
  const float x3 = tf3.xTrackingFrame;
  const float y3 = tf3.positionTrackingFrame[0];
  const float z3 = tf3.positionTrackingFrame[1];

  const bool zeroField{o2::gpu::CAMath::Abs(bz) < o2::constants::math::Almost0};
  const float tgp = zeroField ? o2::gpu::CAMath::ATan2(y3 - y1, x3 - x1) : 1.f;
  const float crv = zeroField ? 1.f : math_utils::computeCurvature(x3, y3, x2, y2, x1, y1);
  const float snp = zeroField ? tgp / o2::gpu::CAMath::Sqrt(1.f + tgp * tgp) : crv * (x3 - math_utils::computeCurvatureCentreX(x3, y3, x2, y2, x1, y1));
  const float tgl12 = math_utils::computeTanDipAngle(x1, y1, x2, y2, z1, z2);
  const float tgl23 = math_utils::computeTanDipAngle(x2, y2, x3, y3, z2, z3);
  const float q2pt = zeroField ? 1.f / o2::track::kMostProbablePt : crv / (bz * o2::constants::math::B2C);
  const float q2pt2 = crv * crv;
  const float sg2q2pt = o2::track::kC1Pt2max * (q2pt2 > 0.0005 ? (q2pt2 < 1 ? q2pt2 : 1) : 0.0005);
  return track::TrackParCov(tf3.xTrackingFrame, tf3.alphaTrackingFrame,
                            {y3, z3, snp, 0.5f * (tgl12 + tgl23), q2pt},
                            {tf3.covarianceTrackingFrame[0],
                             tf3.covarianceTrackingFrame[1], tf3.covarianceTrackingFrame[2],
                             0.f, 0.f, track::kCSnp2max,
                             0.f, 0.f, 0.f, track::kCTgl2max,
                             0.f, 0.f, 0.f, 0.f, sg2q2pt});
}

struct sort_tracklets {
  GPUhd() bool operator()(const Tracklet& a, const Tracklet& b) { return a.firstClusterIndex < b.firstClusterIndex || (a.firstClusterIndex == b.firstClusterIndex && a.secondClusterIndex < b.secondClusterIndex); }
};

struct equal_tracklets {
  GPUhd() bool operator()(const Tracklet& a, const Tracklet& b) { return a.firstClusterIndex == b.firstClusterIndex && a.secondClusterIndex == b.secondClusterIndex; }
};

template <typename T1, typename T2>
struct pair_to_first : public thrust::unary_function<gpuPair<T1, T2>, T1> {
  GPUhd() int operator()(const gpuPair<T1, T2>& a) const
  {
    return a.first;
  }
};

template <typename T1, typename T2>
struct pair_to_second : public thrust::unary_function<gpuPair<T1, T2>, T2> {
  GPUhd() int operator()(const gpuPair<T1, T2>& a) const
  {
    return a.second;
  }
};

template <typename T1, typename T2>
struct is_invalid_pair {
  GPUhd() bool operator()(const gpuPair<T1, T2>& p) const
  {
    return p.first == -1 && p.second == -1;
  }
};

template <typename T1, typename T2>
struct is_valid_pair {
  GPUhd() bool operator()(const gpuPair<T1, T2>& p) const
  {
    return !(p.first == -1 && p.second == -1);
  }
};

GPUd() gpuSpan<const Vertex> getPrimaryVertices(const int rof,
                                                const int* roframesPV,
                                                const int nROF,
                                                const uint8_t* mask,
                                                const Vertex* vertices)
{
  const int start_pv_id = roframesPV[rof];
  const int stop_rof = rof >= nROF - 1 ? nROF : rof + 1;
  size_t delta = mask[rof] ? roframesPV[stop_rof] - start_pv_id : 0; // return empty span if ROF is excluded
  return gpuSpan<const Vertex>(&vertices[start_pv_id], delta);
};

GPUd() gpuSpan<const Cluster> getClustersOnLayer(const int rof,
                                                 const int totROFs,
                                                 const int layer,
                                                 const int** roframesClus,
                                                 const Cluster** clusters)
{
  if (rof < 0 || rof >= totROFs) {
    return gpuSpan<const Cluster>();
  }
  const int start_clus_id{roframesClus[layer][rof]};
  const int stop_rof = rof >= totROFs - 1 ? totROFs : rof + 1;
  const unsigned int delta = roframesClus[layer][stop_rof] - start_clus_id;
  return gpuSpan<const Cluster>(&(clusters[layer][start_clus_id]), delta);
}

template <int nLayers>
GPUg() void fitTrackSeedsKernel(
  CellSeed* trackSeeds,
  const TrackingFrameInfo** foundTrackingFrameInfo,
  o2::its::TrackITSExt* tracks,
  const float* minPts,
  const unsigned int nSeeds,
  const float bz,
  const int startLevel,
  float maxChi2ClusterAttachment,
  float maxChi2NDF,
  const o2::base::Propagator* propagator,
  const o2::base::PropagatorF::MatCorrType matCorrType)
{
  for (int iCurrentTrackSeedIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentTrackSeedIndex < nSeeds; iCurrentTrackSeedIndex += blockDim.x * gridDim.x) {
    auto& seed = trackSeeds[iCurrentTrackSeedIndex];

    TrackITSExt temporaryTrack{seed};

    temporaryTrack.resetCovariance();
    temporaryTrack.setChi2(0);
    int* clusters = seed.getClusters();
    for (int iL{0}; iL < 7; ++iL) {
      temporaryTrack.setExternalClusterIndex(iL, clusters[iL], clusters[iL] != constants::its::UnusedIndex);
    }
    bool fitSuccess = fitTrack(temporaryTrack,               // TrackITSExt& track,
                               0,                            // int lastLayer,
                               nLayers,                      // int firstLayer,
                               1,                            // int firstCluster,
                               maxChi2ClusterAttachment,     // float maxChi2ClusterAttachment,
                               maxChi2NDF,                   // float maxChi2NDF,
                               o2::constants::math::VeryBig, // float maxQoverPt,
                               0,                            // nCl,
                               bz,                           // float bz,
                               foundTrackingFrameInfo,       // TrackingFrameInfo** trackingFrameInfo,
                               propagator,                   // const o2::base::Propagator* propagator,
                               matCorrType);                 // o2::base::PropagatorF::MatCorrType matCorrType
    if (!fitSuccess) {
      continue;
    }
    temporaryTrack.getParamOut() = temporaryTrack.getParamIn();
    temporaryTrack.resetCovariance();
    temporaryTrack.setChi2(0);

    fitSuccess = fitTrack(temporaryTrack,           // TrackITSExt& track,
                          nLayers - 1,              // int lastLayer,
                          -1,                       // int firstLayer,
                          -1,                       // int firstCluster,
                          maxChi2ClusterAttachment, // float maxChi2ClusterAttachment,
                          maxChi2NDF,               // float maxChi2NDF,
                          50.f,                     // float maxQoverPt,
                          0,                        // nCl,
                          bz,                       // float bz,
                          foundTrackingFrameInfo,   // TrackingFrameInfo** trackingFrameInfo,
                          propagator,               // const o2::base::Propagator* propagator,
                          matCorrType);             // o2::base::PropagatorF::MatCorrType matCorrType
    if (!fitSuccess || temporaryTrack.getPt() < minPts[nLayers - temporaryTrack.getNClusters()]) {
      continue;
    }
    tracks[iCurrentTrackSeedIndex] = temporaryTrack;
  }
}

template <bool initRun, int nLayers = 7> // Version for new tracker to supersede the old one
GPUg() void computeLayerCellNeighboursKernel(
  CellSeed** cellSeedArray,
  int* neighboursLUT,
  int* neighboursIndexTable,
  int** cellsLUTs,
  gpuPair<int, int>* cellNeighbours,
  const float maxChi2ClusterAttachment,
  const float bz,
  const int layerIndex,
  const unsigned int nCells,
  const int maxCellNeighbours = 1e2)
{
  for (int iCurrentCellIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentCellIndex < nCells; iCurrentCellIndex += blockDim.x * gridDim.x) {
    const auto& currentCellSeed{cellSeedArray[layerIndex][iCurrentCellIndex]};
    const int nextLayerTrackletIndex{currentCellSeed.getSecondTrackletIndex()};
    const int nextLayerFirstCellIndex{cellsLUTs[layerIndex + 1][nextLayerTrackletIndex]};
    const int nextLayerLastCellIndex{cellsLUTs[layerIndex + 1][nextLayerTrackletIndex + 1]};
    int foundNeighbours{0};
    for (int iNextCell{nextLayerFirstCellIndex}; iNextCell < nextLayerLastCellIndex; ++iNextCell) {
      CellSeed nextCellSeed{cellSeedArray[layerIndex + 1][iNextCell]};      // Copy
      if (nextCellSeed.getFirstTrackletIndex() != nextLayerTrackletIndex) { // Check if cells share the same tracklet
        break;
      }
      if (!nextCellSeed.rotate(currentCellSeed.getAlpha()) ||
          !nextCellSeed.propagateTo(currentCellSeed.getX(), bz)) {
        continue;
      }
      float chi2 = currentCellSeed.getPredictedChi2(nextCellSeed);
      if (chi2 > maxChi2ClusterAttachment) /// TODO: switch to the chi2 wrt cluster to avoid correlation
      {
        continue;
      }
      if constexpr (initRun) {
        atomicAdd(neighboursLUT + iNextCell, 1);
        foundNeighbours++;
        neighboursIndexTable[iCurrentCellIndex]++;
      } else {
        cellNeighbours[neighboursIndexTable[iCurrentCellIndex] + foundNeighbours] = {iCurrentCellIndex, iNextCell};
        foundNeighbours++;
        // FIXME: this is prone to race conditions: check on level is not atomic
        const int currentCellLevel{currentCellSeed.getLevel()};
        if (currentCellLevel >= nextCellSeed.getLevel()) {
          cellSeedArray[layerIndex + 1][iNextCell].setLevel(currentCellLevel + 1);
        }
      }
    }
  }
}

template <bool initRun, int nLayers = 7>
GPUg() void computeLayerCellsKernel(
  const Cluster** sortedClusters,
  const Cluster** unsortedClusters,
  const TrackingFrameInfo** tfInfo,
  Tracklet** tracklets,
  int** trackletsLUT,
  const int nTrackletsCurrent,
  const int layer,
  CellSeed* cells,
  int** cellsLUTs,
  const float bz,
  const float maxChi2ClusterAttachment,
  const float cellDeltaTanLambdaSigma,
  const float nSigmaCut)
{
  constexpr float radl = 9.36f;                                                           // Radiation length of Si [cm].
  constexpr float rho = 2.33f;                                                            // Density of Si [g/cm^3].
  constexpr float layerxX0[7] = {5.e-3f, 5.e-3f, 5.e-3f, 1.e-2f, 1.e-2f, 1.e-2f, 1.e-2f}; // Hardcoded here for the moment.
  for (int iCurrentTrackletIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentTrackletIndex < nTrackletsCurrent; iCurrentTrackletIndex += blockDim.x * gridDim.x) {
    const Tracklet& currentTracklet = tracklets[layer][iCurrentTrackletIndex];
    const int nextLayerClusterIndex{currentTracklet.secondClusterIndex};
    const int nextLayerFirstTrackletIndex{trackletsLUT[layer + 1][nextLayerClusterIndex]};
    const int nextLayerLastTrackletIndex{trackletsLUT[layer + 1][nextLayerClusterIndex + 1]};
    if (nextLayerFirstTrackletIndex == nextLayerLastTrackletIndex) {
      continue;
    }
    int foundCells{0};
    for (int iNextTrackletIndex{nextLayerFirstTrackletIndex}; iNextTrackletIndex < nextLayerLastTrackletIndex; ++iNextTrackletIndex) {
      if (tracklets[layer + 1][iNextTrackletIndex].firstClusterIndex != nextLayerClusterIndex) {
        break;
      }
      const Tracklet& nextTracklet = tracklets[layer + 1][iNextTrackletIndex];
      const float deltaTanLambda{o2::gpu::CAMath::Abs(currentTracklet.tanLambda - nextTracklet.tanLambda)};

      if (deltaTanLambda / cellDeltaTanLambdaSigma < nSigmaCut) {
        const int clusId[3]{
          sortedClusters[layer][currentTracklet.firstClusterIndex].clusterId,
          sortedClusters[layer + 1][nextTracklet.firstClusterIndex].clusterId,
          sortedClusters[layer + 2][nextTracklet.secondClusterIndex].clusterId};

        const auto& cluster1_glo = unsortedClusters[layer][clusId[0]];
        const auto& cluster2_glo = unsortedClusters[layer + 1][clusId[1]];
        const auto& cluster3_tf = tfInfo[layer + 2][clusId[2]];
        auto track{buildTrackSeed(cluster1_glo, cluster2_glo, cluster3_tf, bz)};
        float chi2{0.f};
        bool good{false};
        for (int iC{2}; iC--;) {
          const TrackingFrameInfo& trackingHit = tfInfo[layer + iC][clusId[iC]];
          if (!track.rotate(trackingHit.alphaTrackingFrame)) {
            break;
          }
          if (!track.propagateTo(trackingHit.xTrackingFrame, bz)) {
            break;
          }

          if (!track.correctForMaterial(layerxX0[layer + iC], layerxX0[layer] * radl * rho, true)) {
            break;
          }

          const auto predChi2{track.getPredictedChi2Quiet(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)};
          if (!track.o2::track::TrackParCov::update(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)) {
            break;
          }
          if (!iC && predChi2 > maxChi2ClusterAttachment) {
            break;
          }
          good = !iC;
          chi2 += predChi2;
        }
        if (!good) {
          continue;
        }
        if constexpr (!initRun) {
          new (cells + cellsLUTs[layer][iCurrentTrackletIndex] + foundCells) CellSeed{layer, clusId[0], clusId[1], clusId[2], iCurrentTrackletIndex, iNextTrackletIndex, track, chi2};
        }
        ++foundCells;
        if constexpr (initRun) {
          cellsLUTs[layer][iCurrentTrackletIndex] = foundCells;
        }
      }
    }
  }
}

template <bool initRun = true, int nLayers = 7>
GPUg() void computeLayerTrackletsMultiROFKernel(
  const IndexTableUtils* utils,
  const uint8_t* multMask,
  const int layerIndex,
  const int startROF,
  const int endROF,
  const int totalROFs,
  const int deltaROF,
  const Vertex* vertices,
  const int* rofPV,
  const int nVertices,
  const int vertexId,
  const Cluster** clusters,           // Input data rof0
  const int** ROFClusters,            // Number of clusters on layers per ROF
  const unsigned char** usedClusters, // Used clusters
  const int** indexTables,            // Input data rof0-delta <rof0< rof0+delta (up to 3 rofs)
  Tracklet** tracklets,               // Output data
  int** trackletsLUT,
  const int iteration,
  const float NSigmaCut,
  const float phiCut,
  const float resolutionPV,
  const float minR,
  const float maxR,
  const float positionResolution,
  const float meanDeltaR = -42.f,
  const float MSAngle = -42.f)
{
  const int phiBins{utils->getNphiBins()};
  const int zBins{utils->getNzBins()};
  for (unsigned int iROF{blockIdx.x}; iROF < endROF - startROF; iROF += gridDim.x) {
    const short rof0 = iROF + startROF;
    auto primaryVertices = getPrimaryVertices(rof0, rofPV, totalROFs, multMask, vertices);
    const auto startVtx{vertexId >= 0 ? vertexId : 0};
    const auto endVtx{vertexId >= 0 ? o2::gpu::CAMath::Min(vertexId + 1, static_cast<int>(primaryVertices.size())) : static_cast<int>(primaryVertices.size())};
    const short minROF = o2::gpu::CAMath::Max(startROF, static_cast<int>(rof0 - deltaROF));
    const short maxROF = o2::gpu::CAMath::Min(endROF - 1, static_cast<int>(rof0 + deltaROF));
    auto clustersCurrentLayer = getClustersOnLayer(rof0, totalROFs, layerIndex, ROFClusters, clusters);
    if (clustersCurrentLayer.empty()) {
      continue;
    }

    for (int currentClusterIndex = threadIdx.x; currentClusterIndex < clustersCurrentLayer.size(); currentClusterIndex += blockDim.x) {
      unsigned int storedTracklets{0};
      auto currentCluster{clustersCurrentLayer[currentClusterIndex]};
      const int currentSortedIndex{ROFClusters[layerIndex][rof0] + currentClusterIndex};
      if (usedClusters[layerIndex][currentCluster.clusterId]) {
        continue;
      }

      const float inverseR0{1.f / currentCluster.radius};
      for (int iV{startVtx}; iV < endVtx; ++iV) {
        auto& primaryVertex{primaryVertices[iV]};
        if (primaryVertex.isFlagSet(2) && iteration != 3) {
          continue;
        }
        const float resolution = o2::gpu::CAMath::Sqrt(Sq(resolutionPV) / primaryVertex.getNContributors() + Sq(positionResolution));
        const float tanLambda{(currentCluster.zCoordinate - primaryVertex.getZ()) * inverseR0};
        const float zAtRmin{tanLambda * (minR - currentCluster.radius) + currentCluster.zCoordinate};
        const float zAtRmax{tanLambda * (maxR - currentCluster.radius) + currentCluster.zCoordinate};
        const float sqInverseDeltaZ0{1.f / (Sq(currentCluster.zCoordinate - primaryVertex.getZ()) + 2.e-8f)}; /// protecting from overflows adding the detector resolution
        const float sigmaZ{o2::gpu::CAMath::Sqrt(Sq(resolution) * Sq(tanLambda) * ((Sq(inverseR0) + sqInverseDeltaZ0) * Sq(meanDeltaR) + 1.f) + Sq(meanDeltaR * MSAngle))};
        const int4 selectedBinsRect{getBinsRect(currentCluster, layerIndex, *utils, zAtRmin, zAtRmax, sigmaZ * NSigmaCut, phiCut)};
        if (selectedBinsRect.x == 0 && selectedBinsRect.y == 0 && selectedBinsRect.z == 0 && selectedBinsRect.w == 0) {
          continue;
        }
        int phiBinsNum{selectedBinsRect.w - selectedBinsRect.y + 1};

        if (phiBinsNum < 0) {
          phiBinsNum += phiBins;
        }

        const int tableSize{phiBins * zBins + 1};
        for (short rof1{minROF}; rof1 <= maxROF; ++rof1) {
          auto clustersNextLayer = getClustersOnLayer(rof1, totalROFs, layerIndex + 1, ROFClusters, clusters);
          if (clustersNextLayer.empty()) {
            continue;
          }
          for (int iPhiCount{0}; iPhiCount < phiBinsNum; iPhiCount++) {
            int iPhiBin = (selectedBinsRect.y + iPhiCount) % phiBins;
            const int firstBinIndex{utils->getBinIndex(selectedBinsRect.x, iPhiBin)};
            const int maxBinIndex{firstBinIndex + selectedBinsRect.z - selectedBinsRect.x + 1};
            const int firstRowClusterIndex = indexTables[layerIndex + 1][(rof1 - startROF) * tableSize + firstBinIndex];
            const int maxRowClusterIndex = indexTables[layerIndex + 1][(rof1 - startROF) * tableSize + maxBinIndex];
            for (int nextClusterIndex{firstRowClusterIndex}; nextClusterIndex < maxRowClusterIndex; ++nextClusterIndex) {
              if (nextClusterIndex >= clustersNextLayer.size()) {
                break;
              }
              const Cluster& nextCluster{clustersNextLayer[nextClusterIndex]};
              if (usedClusters[layerIndex + 1][nextCluster.clusterId]) {
                continue;
              }
              const float deltaPhi{o2::gpu::CAMath::Abs(currentCluster.phi - nextCluster.phi)};
              const float deltaZ{o2::gpu::CAMath::Abs(tanLambda * (nextCluster.radius - currentCluster.radius) + currentCluster.zCoordinate - nextCluster.zCoordinate)};
              const int nextSortedIndex{ROFClusters[layerIndex + 1][rof1] + nextClusterIndex};
              if (deltaZ / sigmaZ < NSigmaCut && (deltaPhi < phiCut || o2::gpu::CAMath::Abs(deltaPhi - constants::math::TwoPi) < phiCut)) {
                if constexpr (initRun) {
                  trackletsLUT[layerIndex][currentSortedIndex]++; // we need l0 as well for usual exclusive sums.
                } else {
                  const float phi{o2::gpu::CAMath::ATan2(currentCluster.yCoordinate - nextCluster.yCoordinate, currentCluster.xCoordinate - nextCluster.xCoordinate)};
                  const float tanL{(currentCluster.zCoordinate - nextCluster.zCoordinate) / (currentCluster.radius - nextCluster.radius)};
                  new (tracklets[layerIndex] + trackletsLUT[layerIndex][currentSortedIndex] + storedTracklets) Tracklet{currentSortedIndex, nextSortedIndex, tanL, phi, rof0, rof1};
                }
                ++storedTracklets;
              }
            }
          }
        }
      }
    }
  }
}

template <int nLayers = 7>
GPUg() void compileTrackletsLookupTableKernel(const Tracklet* tracklets,
                                              int* trackletsLookUpTable,
                                              const int nTracklets)
{
  for (int currentTrackletIndex = blockIdx.x * blockDim.x + threadIdx.x; currentTrackletIndex < nTracklets; currentTrackletIndex += blockDim.x * gridDim.x) {
    atomicAdd(&trackletsLookUpTable[tracklets[currentTrackletIndex].firstClusterIndex], 1);
  }
}

template <bool dryRun, bool debug = false, int nLayers = 7>
GPUg() void processNeighboursKernel(const int layer,
                                    const int level,
                                    CellSeed** allCellSeeds,
                                    CellSeed* currentCellSeeds,
                                    const int* currentCellIds,
                                    const unsigned int nCurrentCells,
                                    CellSeed* updatedCellSeeds,
                                    int* updatedCellsIds,
                                    int* foundSeedsTable,               // auxiliary only in GPU code to compute the number of cells per iteration
                                    const unsigned char** usedClusters, // Used clusters
                                    int* neighbours,
                                    int* neighboursLUT,
                                    const TrackingFrameInfo** foundTrackingFrameInfo,
                                    const float bz,
                                    const float MaxChi2ClusterAttachment,
                                    const o2::base::Propagator* propagator,
                                    const o2::base::PropagatorF::MatCorrType matCorrType)
{
  constexpr float radl = 9.36f;                                                           // Radiation length of Si [cm].
  constexpr float rho = 2.33f;                                                            // Density of Si [g/cm^3].
  constexpr float layerxX0[7] = {5.e-3f, 5.e-3f, 5.e-3f, 1.e-2f, 1.e-2f, 1.e-2f, 1.e-2f}; // Hardcoded here for the moment.
  for (unsigned int iCurrentCell = blockIdx.x * blockDim.x + threadIdx.x; iCurrentCell < nCurrentCells; iCurrentCell += blockDim.x * gridDim.x) {
    int foundSeeds{0};
    const auto& currentCell{currentCellSeeds[iCurrentCell]};
    if (currentCell.getLevel() != level) {
      continue;
    }
    if (currentCellIds == nullptr && (usedClusters[layer][currentCell.getFirstClusterIndex()] ||
                                      usedClusters[layer + 1][currentCell.getSecondClusterIndex()] ||
                                      usedClusters[layer + 2][currentCell.getThirdClusterIndex()])) {
      continue;
    }
    const int cellId = currentCellIds == nullptr ? iCurrentCell : currentCellIds[iCurrentCell];

    const int startNeighbourId{cellId ? neighboursLUT[cellId - 1] : 0};
    const int endNeighbourId{neighboursLUT[cellId]};

    for (int iNeighbourCell{startNeighbourId}; iNeighbourCell < endNeighbourId; ++iNeighbourCell) {
      const int neighbourCellId = neighbours[iNeighbourCell];
      const CellSeed& neighbourCell = allCellSeeds[layer - 1][neighbourCellId];

      if (neighbourCell.getSecondTrackletIndex() != currentCell.getFirstTrackletIndex()) {
        continue;
      }
      if (usedClusters[layer - 1][neighbourCell.getFirstClusterIndex()]) {
        continue;
      }
      if (currentCell.getLevel() - 1 != neighbourCell.getLevel()) {
        continue;
      }
      CellSeed seed{currentCell};
      auto& trHit = foundTrackingFrameInfo[layer - 1][neighbourCell.getFirstClusterIndex()];

      if (!seed.rotate(trHit.alphaTrackingFrame)) {
        continue;
      }

      if (!propagator->propagateToX(seed, trHit.xTrackingFrame, bz, o2::base::PropagatorImpl<float>::MAX_SIN_PHI, o2::base::PropagatorImpl<float>::MAX_STEP, matCorrType)) {
        continue;
      }

      if (matCorrType == o2::base::PropagatorF::MatCorrType::USEMatCorrNONE) {
        if (!seed.correctForMaterial(layerxX0[layer - 1], layerxX0[layer - 1] * radl * rho, true)) {
          continue;
        }
      }

      auto predChi2{seed.getPredictedChi2Quiet(trHit.positionTrackingFrame, trHit.covarianceTrackingFrame)};
      if ((predChi2 > MaxChi2ClusterAttachment) || predChi2 < 0.f) {
        continue;
      }
      seed.setChi2(seed.getChi2() + predChi2);
      if (!seed.o2::track::TrackParCov::update(trHit.positionTrackingFrame, trHit.covarianceTrackingFrame)) {
        continue;
      }
      seed.getClusters()[layer - 1] = neighbourCell.getFirstClusterIndex();
      seed.setLevel(neighbourCell.getLevel());
      seed.setFirstTrackletIndex(neighbourCell.getFirstTrackletIndex());
      seed.setSecondTrackletIndex(neighbourCell.getSecondTrackletIndex());
      if constexpr (dryRun) {
        foundSeedsTable[iCurrentCell]++;
      } else {
        updatedCellsIds[foundSeedsTable[iCurrentCell] + foundSeeds] = neighbourCellId;
        updatedCellSeeds[foundSeedsTable[iCurrentCell] + foundSeeds] = seed;
      }
      foundSeeds++;
    }
  }
}

/////////////////////////////////////////
// Debug Kernels
/////////////////////////////////////////

template <typename T>
GPUd() void pPointer(T* ptr)
{
  printf("[%p]\t", ptr);
}

template <typename... Args>
GPUg() void printPointersKernel(std::tuple<Args...> args)
{
  auto print_all = [&](auto... ptrs) {
    (pPointer(ptrs), ...);
  };
  std::apply(print_all, args);
}

template <typename T>
struct trackletSortEmptyFunctor : public thrust::binary_function<T, T, bool> {
  GPUhd() bool operator()(const T& lhs, const T& rhs) const
  {
    return lhs.firstClusterIndex > rhs.firstClusterIndex;
  }
};

template <typename T>
struct trackletSortIndexFunctor : public thrust::binary_function<T, T, bool> {
  GPUhd() bool operator()(const T& lhs, const T& rhs) const
  {
    return lhs.firstClusterIndex < rhs.firstClusterIndex || (lhs.firstClusterIndex == rhs.firstClusterIndex && lhs.secondClusterIndex < rhs.secondClusterIndex);
  }
};

GPUg() void printBufferLayerOnThread(const int layer, const int* v, unsigned int size, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < size; ++i) {
      if (!(i % len)) {
        printf("\n layer %d: ===> %d/%d\t", layer, i, (int)size);
      }
      printf("%d\t", v[i]);
    }
    printf("\n");
  }
}

GPUg() void printMatrixRow(const int row, int** mat, const unsigned int rowLength, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < rowLength; ++i) {
      if (!(i % len)) {
        printf("\n row %d: ===> %d/%d\t", row, i, (int)rowLength);
      }
      printf("%d\t", mat[row][i]);
    }
    printf("\n");
  }
}

GPUg() void printBufferPointersLayerOnThread(const int layer, void** v, unsigned int size, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < size; ++i) {
      if (!(i % len)) {
        printf("\n layer %d: ===> %d/%d\t", layer, i, (int)size);
      }
      printf("%p\t", (void*)v[i]);
    }
    printf("\n");
  }
}

GPUg() void printVertices(const Vertex* v, unsigned int size, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    printf("vertices: \n");
    for (int i{0}; i < size; ++i) {
      printf("\tx=%f y=%f z=%f\n", v[i].getX(), v[i].getY(), v[i].getZ());
    }
  }
}

GPUg() void printNeighbours(const gpuPair<int, int>* neighbours,
                            const int* nNeighboursIndexTable,
                            const unsigned int nCells,
                            const unsigned int tId = 0)
{
  for (unsigned int iNeighbour{0}; iNeighbour < nNeighboursIndexTable[nCells]; ++iNeighbour) {
    if (threadIdx.x == tId) {
      printf("%d -> %d\n", neighbours[iNeighbour].first, neighbours[iNeighbour].second);
    }
  }
}

GPUg() void printTrackletsLUTPerROF(const int layerId,
                                    const int** ROFClusters,
                                    int** luts,
                                    const int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (auto rofId{0}; rofId < 2304; ++rofId) {
      int nClus = ROFClusters[layerId][rofId + 1] - ROFClusters[layerId][rofId];
      if (!nClus) {
        continue;
      }
      printf("rof: %d (%d) ==> ", rofId, nClus);

      for (int iC{0}; iC < nClus; ++iC) {
        int nT = luts[layerId][ROFClusters[layerId][rofId] + iC];
        printf("%d\t", nT);
      }
      printf("\n");
    }
  }
}

GPUg() void printCellSeeds(CellSeed* seed, int nCells, const unsigned int tId = 0)
{
  for (unsigned int iCell{0}; iCell < nCells; ++iCell) {
    if (threadIdx.x == tId) {
      seed[iCell].printCell();
    }
  }
}
} // namespace gpu

template <int nLayers>
void countTrackletsInROFsHandler(const IndexTableUtils* utils,
                                 const uint8_t* multMask,
                                 const int startROF,
                                 const int endROF,
                                 const int maxROF,
                                 const int deltaROF,
                                 const int vertexId,
                                 const Vertex* vertices,
                                 const int* rofPV,
                                 const int nVertices,
                                 const Cluster** clusters,
                                 std::vector<unsigned int> nClusters,
                                 const int** ROFClusters,
                                 const unsigned char** usedClusters,
                                 const int** clustersIndexTables,
                                 int** trackletsLUTs,
                                 gsl::span<int*> trackletsLUTsHost,
                                 const int iteration,
                                 const float NSigmaCut,
                                 std::vector<float>& phiCuts,
                                 const float resolutionPV,
                                 std::vector<float>& minRs,
                                 std::vector<float>& maxRs,
                                 std::vector<float>& resolutions,
                                 std::vector<float>& radii,
                                 std::vector<float>& mulScatAng,
                                 const int nBlocks,
                                 const int nThreads)
{
  for (int iLayer = 0; iLayer < nLayers - 1; ++iLayer) {
    gpu::computeLayerTrackletsMultiROFKernel<true><<<nBlocks, nThreads>>>(
      utils,
      multMask,
      iLayer,
      startROF,
      endROF,
      maxROF,
      deltaROF,
      vertices,
      rofPV,
      nVertices,
      vertexId,
      clusters,
      ROFClusters,
      usedClusters,
      clustersIndexTables,
      nullptr,
      trackletsLUTs,
      iteration,
      NSigmaCut,
      phiCuts[iLayer],
      resolutionPV,
      minRs[iLayer + 1],
      maxRs[iLayer + 1],
      resolutions[iLayer],
      radii[iLayer + 1] - radii[iLayer],
      mulScatAng[iLayer]);
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,            // d_temp_storage
                                                temp_storage_bytes,        // temp_storage_bytes
                                                trackletsLUTsHost[iLayer], // d_in
                                                trackletsLUTsHost[iLayer], // d_out
                                                nClusters[iLayer] + 1,     // num_items
                                                0));                       // NOLINT: this is the offset of the sum, not a pointer
    discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
    gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,            // d_temp_storage
                                                temp_storage_bytes,        // temp_storage_bytes
                                                trackletsLUTsHost[iLayer], // d_in
                                                trackletsLUTsHost[iLayer], // d_out
                                                nClusters[iLayer] + 1,     // num_items
                                                0));                       // NOLINT: this is the offset of the sum, not a pointer
    gpuCheckError(hipFree(d_temp_storage));
  }
}

template <int nLayers>
void computeTrackletsInROFsHandler(const IndexTableUtils* utils,
                                   const uint8_t* multMask,
                                   const int startROF,
                                   const int endROF,
                                   const int maxROF,
                                   const int deltaROF,
                                   const int vertexId,
                                   const Vertex* vertices,
                                   const int* rofPV,
                                   const int nVertices,
                                   const Cluster** clusters,
                                   std::vector<unsigned int> nClusters,
                                   const int** ROFClusters,
                                   const unsigned char** usedClusters,
                                   const int** clustersIndexTables,
                                   Tracklet** tracklets,
                                   gsl::span<Tracklet*> spanTracklets,
                                   gsl::span<int> nTracklets,
                                   int** trackletsLUTs,
                                   gsl::span<int*> trackletsLUTsHost,
                                   const int iteration,
                                   const float NSigmaCut,
                                   std::vector<float>& phiCuts,
                                   const float resolutionPV,
                                   std::vector<float>& minRs,
                                   std::vector<float>& maxRs,
                                   std::vector<float>& resolutions,
                                   std::vector<float>& radii,
                                   std::vector<float>& mulScatAng,
                                   const int nBlocks,
                                   const int nThreads)
{
  for (int iLayer = 0; iLayer < nLayers - 1; ++iLayer) {
    gpu::computeLayerTrackletsMultiROFKernel<false><<<nBlocks, nThreads>>>(utils,
                                                                           multMask,
                                                                           iLayer,
                                                                           startROF,
                                                                           endROF,
                                                                           maxROF,
                                                                           deltaROF,
                                                                           vertices,
                                                                           rofPV,
                                                                           nVertices,
                                                                           vertexId,
                                                                           clusters,
                                                                           ROFClusters,
                                                                           usedClusters,
                                                                           clustersIndexTables,
                                                                           tracklets,
                                                                           trackletsLUTs,
                                                                           iteration,
                                                                           NSigmaCut,
                                                                           phiCuts[iLayer],
                                                                           resolutionPV,
                                                                           minRs[iLayer + 1],
                                                                           maxRs[iLayer + 1],
                                                                           resolutions[iLayer],
                                                                           radii[iLayer + 1] - radii[iLayer],
                                                                           mulScatAng[iLayer]);
    thrust::device_ptr<Tracklet> tracklets_ptr(spanTracklets[iLayer]);
    thrust::sort(thrust::device, tracklets_ptr, tracklets_ptr + nTracklets[iLayer], gpu::sort_tracklets());
    auto unique_end = thrust::unique(thrust::device, tracklets_ptr, tracklets_ptr + nTracklets[iLayer], gpu::equal_tracklets());
    nTracklets[iLayer] = unique_end - tracklets_ptr;
    if (iLayer > 0) {
      gpuCheckError(hipMemset(trackletsLUTsHost[iLayer], 0, nClusters[iLayer] * sizeof(int)));
      gpu::compileTrackletsLookupTableKernel<<<nBlocks, nThreads>>>(spanTracklets[iLayer], trackletsLUTsHost[iLayer], nTracklets[iLayer]);
      void* d_temp_storage = nullptr;
      size_t temp_storage_bytes = 0;
      gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,            // d_temp_storage
                                                  temp_storage_bytes,        // temp_storage_bytes
                                                  trackletsLUTsHost[iLayer], // d_in
                                                  trackletsLUTsHost[iLayer], // d_out
                                                  nClusters[iLayer] + 1,     // num_items
                                                  0));                       // NOLINT: this is the offset of the sum, not a pointer
      discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
      gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,            // d_temp_storage
                                                  temp_storage_bytes,        // temp_storage_bytes
                                                  trackletsLUTsHost[iLayer], // d_in
                                                  trackletsLUTsHost[iLayer], // d_out
                                                  nClusters[iLayer] + 1,     // num_items
                                                  0));                       // NOLINT: this is the offset of the sum, not a pointer
      gpuCheckError(hipFree(d_temp_storage));
    }
  }
}

void countCellsHandler(
  const Cluster** sortedClusters,
  const Cluster** unsortedClusters,
  const TrackingFrameInfo** tfInfo,
  Tracklet** tracklets,
  int** trackletsLUT,
  const int nTracklets,
  const int layer,
  CellSeed* cells,
  int** cellsLUTsArrayDevice,
  int* cellsLUTsHost,
  const float bz,
  const float maxChi2ClusterAttachment,
  const float cellDeltaTanLambdaSigma,
  const float nSigmaCut,
  const int nBlocks,
  const int nThreads)
{
  gpu::computeLayerCellsKernel<true><<<nBlocks, nThreads>>>(
    sortedClusters,           // const Cluster**
    unsortedClusters,         // const Cluster**
    tfInfo,                   // const TrackingFrameInfo**
    tracklets,                // const Tracklets**
    trackletsLUT,             // const int**
    nTracklets,               // const int
    layer,                    // const int
    cells,                    // CellSeed*
    cellsLUTsArrayDevice,     // int**
    bz,                       // const float
    maxChi2ClusterAttachment, // const float
    cellDeltaTanLambdaSigma,  // const float
    nSigmaCut);               // const float
  void* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,     // d_temp_storage
                                              temp_storage_bytes, // temp_storage_bytes
                                              cellsLUTsHost,      // d_in
                                              cellsLUTsHost,      // d_out
                                              nTracklets + 1,     // num_items
                                              0));                // NOLINT: this is the offset of the sum, not a pointer
  discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,     // d_temp_storage
                                              temp_storage_bytes, // temp_storage_bytes
                                              cellsLUTsHost,      // d_in
                                              cellsLUTsHost,      // d_out
                                              nTracklets + 1,     // num_items
                                              0));                // NOLINT: this is the offset of the sum, not a pointer
  gpuCheckError(hipFree(d_temp_storage));
}

void computeCellsHandler(
  const Cluster** sortedClusters,
  const Cluster** unsortedClusters,
  const TrackingFrameInfo** tfInfo,
  Tracklet** tracklets,
  int** trackletsLUT,
  const int nTracklets,
  const int layer,
  CellSeed* cells,
  int** cellsLUTsArrayDevice,
  int* cellsLUTsHost,
  const float bz,
  const float maxChi2ClusterAttachment,
  const float cellDeltaTanLambdaSigma,
  const float nSigmaCut,
  const int nBlocks,
  const int nThreads)
{
  gpu::computeLayerCellsKernel<false><<<nBlocks, nThreads>>>(
    sortedClusters,           // const Cluster**
    unsortedClusters,         // const Cluster**
    tfInfo,                   // const TrackingFrameInfo**
    tracklets,                // const Tracklets**
    trackletsLUT,             // const int**
    nTracklets,               // const int
    layer,                    // const int
    cells,                    // CellSeed*
    cellsLUTsArrayDevice,     // int**
    bz,                       // const float
    maxChi2ClusterAttachment, // const float
    cellDeltaTanLambdaSigma,  // const float
    nSigmaCut);               // const float
}

void countCellNeighboursHandler(CellSeed** cellsLayersDevice,
                                int* neighboursLUT,
                                int** cellsLUTs,
                                gpuPair<int, int>* cellNeighbours,
                                int* neighboursIndexTable,
                                const float maxChi2ClusterAttachment,
                                const float bz,
                                const int layerIndex,
                                const unsigned int nCells,
                                const unsigned int nCellsNext,
                                const int maxCellNeighbours,
                                const int nBlocks,
                                const int nThreads)
{
  gpu::computeLayerCellNeighboursKernel<true><<<nBlocks, nThreads>>>(
    cellsLayersDevice,
    neighboursLUT,
    neighboursIndexTable,
    cellsLUTs,
    cellNeighbours,
    maxChi2ClusterAttachment,
    bz,
    layerIndex,
    nCells,
    maxCellNeighbours);
  // gpuCheckError(hipPeekAtLastError());
  // gpuCheckError(hipDeviceSynchronize());
  void *d_temp_storage = nullptr, *d_temp_storage_2 = nullptr;
  size_t temp_storage_bytes = 0, temp_storage_bytes_2 = 0;
  gpuCheckError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,     // d_temp_storage
                                              temp_storage_bytes, // temp_storage_bytes
                                              neighboursLUT,      // d_in
                                              neighboursLUT,      // d_out
                                              nCellsNext));       // num_items

  discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
  gpuCheckError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,       // d_temp_storage
                                              temp_storage_bytes,   // temp_storage_bytes
                                              neighboursLUT,        // d_in
                                              neighboursLUT,        // d_out
                                              nCellsNext));         // num_items
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage_2,     // d_temp_storage
                                              temp_storage_bytes_2, // temp_storage_bytes
                                              neighboursIndexTable, // d_in
                                              neighboursIndexTable, // d_out
                                              nCells + 1,           // num_items
                                              0));                  // NOLINT: this is the offset of the sum, not a pointer
  discardResult(hipMalloc(&d_temp_storage_2, temp_storage_bytes_2));
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage_2,     // d_temp_storage
                                              temp_storage_bytes_2, // temp_storage_bytes
                                              neighboursIndexTable, // d_in
                                              neighboursIndexTable, // d_out
                                              nCells + 1,           // num_items
                                              0));                  // NOLINT: this is the offset of the sum, not a pointer
  gpuCheckError(hipFree(d_temp_storage));
  gpuCheckError(hipFree(d_temp_storage_2));
  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

void computeCellNeighboursHandler(CellSeed** cellsLayersDevice,
                                  int* neighboursLUT,
                                  int** cellsLUTs,
                                  gpuPair<int, int>* cellNeighbours,
                                  int* neighboursIndexTable,
                                  const float maxChi2ClusterAttachment,
                                  const float bz,
                                  const int layerIndex,
                                  const unsigned int nCells,
                                  const unsigned int nCellsNext,
                                  const int maxCellNeighbours,
                                  const int nBlocks,
                                  const int nThreads)
{

  gpu::computeLayerCellNeighboursKernel<false><<<o2::gpu::CAMath::Min(nBlocks, GPU_BLOCKS),
                                                 o2::gpu::CAMath::Min(nThreads, GPU_THREADS)>>>(
    cellsLayersDevice,
    neighboursLUT,
    neighboursIndexTable,
    cellsLUTs,
    cellNeighbours,
    maxChi2ClusterAttachment,
    bz,
    layerIndex,
    nCells,
    maxCellNeighbours);
  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

int filterCellNeighboursHandler(std::vector<int>& neighHost, // TODO: eventually remove this!
                                gpuPair<int, int>* cellNeighbourPairs,
                                int* cellNeighbours,
                                unsigned int nNeigh)
{
  thrust::device_ptr<gpuPair<int, int>> neighVectorPairs(cellNeighbourPairs);
  thrust::device_ptr<int> validNeighs(cellNeighbours);
  thrust::device_vector<int> keys(nNeigh); // TODO: externally allocate.
  thrust::device_vector<int> vals(nNeigh); // TODO: externally allocate.
  thrust::copy(thrust::make_transform_iterator(neighVectorPairs, gpu::pair_to_second<int, int>()),
               thrust::make_transform_iterator(neighVectorPairs + nNeigh, gpu::pair_to_second<int, int>()),
               keys.begin());
  thrust::sequence(vals.begin(), vals.end());
  thrust::sort_by_key(keys.begin(), keys.end(), vals.begin());
  thrust::device_vector<gpuPair<int, int>> sortedNeigh(nNeigh);
  thrust::copy(thrust::make_permutation_iterator(neighVectorPairs, vals.begin()),
               thrust::make_permutation_iterator(neighVectorPairs, vals.end()),
               sortedNeigh.begin());
  discardResult(hipDeviceSynchronize());
  auto trimmedBegin = thrust::find_if(sortedNeigh.begin(), sortedNeigh.end(), gpu::is_valid_pair<int, int>()); // trim leading -1s
  auto trimmedSize = sortedNeigh.end() - trimmedBegin;
  neighHost.resize(trimmedSize);
  thrust::transform(trimmedBegin, sortedNeigh.end(), validNeighs, gpu::pair_to_first<int, int>());
  gpuCheckError(hipMemcpy(neighHost.data(), cellNeighbours, trimmedSize * sizeof(int), hipMemcpyDeviceToHost));

  return trimmedSize;
}

template <int nLayers>
void processNeighboursHandler(const int startLayer,
                              const int startLevel,
                              CellSeed** allCellSeeds,
                              CellSeed* currentCellSeeds,
                              const unsigned int nCurrentCells,
                              const unsigned char** usedClusters,
                              int* neighbours,
                              gsl::span<int*> neighboursDeviceLUTs,
                              const TrackingFrameInfo** foundTrackingFrameInfo,
                              const float bz,
                              const float MaxChi2ClusterAttachment,
                              const o2::base::Propagator* propagator,
                              const o2::base::PropagatorF::MatCorrType matCorrType,
                              const std::vector<int>& lastCellIdHost,        // temporary host vector
                              const std::vector<CellSeed>& lastCellSeedHost, // temporary host vector
                              std::vector<int>& updatedCellIdHost,           // temporary host vector
                              std::vector<CellSeed>& updatedCellSeedHost,    // temporary host vector
                              const int nBlocks,
                              const int nThreads)
{
  thrust::device_vector<int> foundSeedsTable(nCurrentCells + 1); // Shortcut: device_vector skips central memory management, we are relying on the contingency. TODO: fix this.
  thrust::device_vector<int> lastCellIds(lastCellIdHost);
  thrust::device_vector<CellSeed> lastCellSeed(lastCellSeedHost);
  gpu::processNeighboursKernel<true><<<nBlocks, nThreads>>>(startLayer,
                                                            startLevel,
                                                            allCellSeeds,
                                                            lastCellIdHost.empty() ? currentCellSeeds : thrust::raw_pointer_cast(&lastCellSeed[0]), // lastCellSeeds
                                                            lastCellIdHost.empty() ? nullptr : thrust::raw_pointer_cast(&lastCellIds[0]),           // lastCellIds,
                                                            lastCellIdHost.empty() ? nCurrentCells : lastCellSeedHost.size(),
                                                            nullptr,                                       // updatedCellSeeds,
                                                            nullptr,                                       // updatedCellsIds,
                                                            thrust::raw_pointer_cast(&foundSeedsTable[0]), // auxiliary only in GPU code to compute the number of cells per iteration
                                                            usedClusters,                                  // Used clusters
                                                            neighbours,
                                                            neighboursDeviceLUTs[startLayer - 1],
                                                            foundTrackingFrameInfo,
                                                            bz,
                                                            MaxChi2ClusterAttachment,
                                                            propagator,
                                                            matCorrType);
  void *d_temp_storage = nullptr, *d_temp_storage_2 = nullptr;
  size_t temp_storage_bytes = 0, temp_storage_bytes_2 = 0;
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,                                // d_temp_storage
                                              temp_storage_bytes,                            // temp_storage_bytes
                                              thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_in
                                              thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_out
                                              nCurrentCells + 1,                             // num_items
                                              0));                                           // NOLINT: failure in clang-tidy
  discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,                                // d_temp_storage
                                              temp_storage_bytes,                            // temp_storage_bytes
                                              thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_in
                                              thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_out
                                              nCurrentCells + 1,                             // num_items
                                              0));                                           // NOLINT: failure in clang-tidy

  thrust::device_vector<int> updatedCellIds(foundSeedsTable.back()) /*, lastCellIds(foundSeedsTable.back())*/;
  thrust::device_vector<CellSeed> updatedCellSeeds(foundSeedsTable.back()) /*, lastCellSeeds(foundSeedsTable.back())*/;

  gpu::processNeighboursKernel<false><<<nBlocks, nThreads>>>(startLayer,
                                                             startLevel,
                                                             allCellSeeds,
                                                             lastCellIdHost.empty() ? currentCellSeeds : thrust::raw_pointer_cast(&lastCellSeed[0]), // lastCellSeeds
                                                             lastCellIdHost.empty() ? nullptr : thrust::raw_pointer_cast(&lastCellIds[0]),           // lastCellIds,
                                                             lastCellIdHost.empty() ? nCurrentCells : lastCellSeedHost.size(),
                                                             thrust::raw_pointer_cast(&updatedCellSeeds[0]), // updatedCellSeeds
                                                             thrust::raw_pointer_cast(&updatedCellIds[0]),   // updatedCellsIds
                                                             thrust::raw_pointer_cast(&foundSeedsTable[0]),  // auxiliary only in GPU code to compute the number of cells per iteration
                                                             usedClusters,                                   // Used clusters
                                                             neighbours,
                                                             neighboursDeviceLUTs[startLayer - 1],
                                                             foundTrackingFrameInfo,
                                                             bz,
                                                             MaxChi2ClusterAttachment,
                                                             propagator,
                                                             matCorrType);

  // Temporary copyback to host to validate the kernel
  updatedCellIdHost.resize(updatedCellIds.size());
  updatedCellSeedHost.resize(updatedCellSeeds.size());
  thrust::copy(updatedCellIds.begin(), updatedCellIds.end(), updatedCellIdHost.begin());
  thrust::copy(updatedCellSeeds.begin(), updatedCellSeeds.end(), updatedCellSeedHost.begin());

  // int level = startLevel;
  // for (int iLayer{startLayer - 1}; iLayer > 0 && level > 2; --iLayer) {
  //   --level;
  //   lastCellSeeds.swap(updatedCellSeeds);
  //   lastCellIds.swap(updatedCellIds);
  //   foundSeedsTable.resize(lastCellSeeds.size() + 1);
  //   thrust::fill(foundSeedsTable.begin(), foundSeedsTable.end(), 0);

  //   gpu::processNeighboursKernel<true, false><<<1, 1>>>(iLayer,
  //                                                       level,
  //                                                       allCellSeeds,
  //                                                       thrust::raw_pointer_cast(&lastCellSeeds[0]),
  //                                                       thrust::raw_pointer_cast(&lastCellIds[0]), // currentCellIds,
  //                                                       lastCellSeeds.size(),
  //                                                       nullptr,                                       // updatedCellSeeds,
  //                                                       nullptr,                                       // updatedCellsIds,
  //                                                       thrust::raw_pointer_cast(&foundSeedsTable[0]), // auxiliary only in GPU code to compute the number of cells per iteration
  //                                                       usedClusters,                                  // Used clusters
  //                                                       neighbours,
  //                                                       neighboursDeviceLUTs[iLayer - 1],
  //                                                       foundTrackingFrameInfo,
  //                                                       bz,
  //                                                       MaxChi2ClusterAttachment,
  //                                                       propagator,
  //                                                       matCorrType);

  //   gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage_2,                              // d_temp_storage
  //                                               temp_storage_bytes,                            // temp_storage_bytes
  //                                               thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_in
  //                                               thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_out
  //                                               foundSeedsTable.size(),                        // num_items
  //                                               0));
  //   discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
  //   gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage_2,                              // d_temp_storage
  //                                               temp_storage_bytes_2,                          // temp_storage_bytes
  //                                               thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_in
  //                                               thrust::raw_pointer_cast(&foundSeedsTable[0]), // d_out
  //                                               foundSeedsTable.size(),                        // num_items
  //                                               0));
  //   updatedCellIds.resize(foundSeedsTable.back(), 0);
  //   updatedCellSeeds.resize(foundSeedsTable.back(), CellSeed());

  //   gpu::processNeighboursKernel<false><<<1, 1>>>(iLayer,
  //                                                 level,
  //                                                 allCellSeeds,
  //                                                 thrust::raw_pointer_cast(&lastCellSeeds[0]),
  //                                                 thrust::raw_pointer_cast(&lastCellIds[0]), // currentCellIds,
  //                                                 lastCellSeeds.size(),
  //                                                 thrust::raw_pointer_cast(&updatedCellSeeds[0]), // updatedCellSeeds
  //                                                 thrust::raw_pointer_cast(&updatedCellIds[0]),   // updatedCellsIds
  //                                                 thrust::raw_pointer_cast(&foundSeedsTable[0]),  // auxiliary only in GPU code to compute the number of cells per iteration
  //                                                 usedClusters,                                   // Used clusters
  //                                                 neighbours,
  //                                                 neighboursDeviceLUTs[iLayer - 1],
  //                                                 foundTrackingFrameInfo,
  //                                                 bz,
  //                                                 MaxChi2ClusterAttachment,
  //                                                 propagator,
  //                                                 matCorrType);
  //   gpu::printCellSeeds<<<1, 1>>>(thrust::raw_pointer_cast(&updatedCellSeeds[0]), updatedCellSeeds.size());
  // }

  gpuCheckError(hipFree(d_temp_storage));
  gpuCheckError(hipFree(d_temp_storage_2));
}

void trackSeedHandler(CellSeed* trackSeeds,
                      const TrackingFrameInfo** foundTrackingFrameInfo,
                      o2::its::TrackITSExt* tracks,
                      std::vector<float>& minPtsHost,
                      const unsigned int nSeeds,
                      const float bz,
                      const int startLevel,
                      float maxChi2ClusterAttachment,
                      float maxChi2NDF,
                      const o2::base::Propagator* propagator,
                      const o2::base::PropagatorF::MatCorrType matCorrType,
                      const int nBlocks,
                      const int nThreads)
{
  thrust::device_vector<float> minPts(minPtsHost);
  gpu::fitTrackSeedsKernel<<<nBlocks, nThreads>>>(
    trackSeeds,                           // CellSeed*
    foundTrackingFrameInfo,               // TrackingFrameInfo**
    tracks,                               // TrackITSExt*
    thrust::raw_pointer_cast(&minPts[0]), // const float* minPts,
    nSeeds,                               // const unsigned int
    bz,                                   // const float
    startLevel,                           // const int
    maxChi2ClusterAttachment,             // float
    maxChi2NDF,                           // float
    propagator,                           // const o2::base::Propagator*
    matCorrType);                         // o2::base::PropagatorF::MatCorrType

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

template void countTrackletsInROFsHandler<7>(const IndexTableUtils* utils,
                                             const uint8_t* multMask,
                                             const int startROF,
                                             const int endROF,
                                             const int maxROF,
                                             const int deltaROF,
                                             const int vertexId,
                                             const Vertex* vertices,
                                             const int* rofPV,
                                             const int nVertices,
                                             const Cluster** clusters,
                                             std::vector<unsigned int> nClusters,
                                             const int** ROFClusters,
                                             const unsigned char** usedClusters,
                                             const int** clustersIndexTables,
                                             int** trackletsLUTs,
                                             gsl::span<int*> trackletsLUTsHost,
                                             const int iteration,
                                             const float NSigmaCut,
                                             std::vector<float>& phiCuts,
                                             const float resolutionPV,
                                             std::vector<float>& minRs,
                                             std::vector<float>& maxRs,
                                             std::vector<float>& resolutions,
                                             std::vector<float>& radii,
                                             std::vector<float>& mulScatAng,
                                             const int nBlocks,
                                             const int nThreads);

template void computeTrackletsInROFsHandler<7>(const IndexTableUtils* utils,
                                               const uint8_t* multMask,
                                               const int startROF,
                                               const int endROF,
                                               const int maxROF,
                                               const int deltaROF,
                                               const int vertexId,
                                               const Vertex* vertices,
                                               const int* rofPV,
                                               const int nVertices,
                                               const Cluster** clusters,
                                               std::vector<unsigned int> nClusters,
                                               const int** ROFClusters,
                                               const unsigned char** usedClusters,
                                               const int** clustersIndexTables,
                                               Tracklet** tracklets,
                                               gsl::span<Tracklet*> spanTracklets,
                                               gsl::span<int> nTracklets,
                                               int** trackletsLUTs,
                                               gsl::span<int*> trackletsLUTsHost,
                                               const int iteration,
                                               const float NSigmaCut,
                                               std::vector<float>& phiCuts,
                                               const float resolutionPV,
                                               std::vector<float>& minRs,
                                               std::vector<float>& maxRs,
                                               std::vector<float>& resolutions,
                                               std::vector<float>& radii,
                                               std::vector<float>& mulScatAng,
                                               const int nBlocks,
                                               const int nThreads);

template void processNeighboursHandler<7>(const int startLayer,
                                          const int startLevel,
                                          CellSeed** allCellSeeds,
                                          CellSeed* currentCellSeeds,
                                          const unsigned int nCurrentCells,
                                          const unsigned char** usedClusters,
                                          int* neighbours,
                                          gsl::span<int*> neighboursDeviceLUTs,
                                          const TrackingFrameInfo** foundTrackingFrameInfo,
                                          const float bz,
                                          const float MaxChi2ClusterAttachment,
                                          const o2::base::Propagator* propagator,
                                          const o2::base::PropagatorF::MatCorrType matCorrType,
                                          const std::vector<int>& lastCellIdHost,        // temporary host vector
                                          const std::vector<CellSeed>& lastCellSeedHost, // temporary host vector
                                          std::vector<int>& updatedCellIdHost,           // temporary host vector
                                          std::vector<CellSeed>& updatedCellSeedHost,    // temporary host vector
                                          const int nBlocks,
                                          const int nThreads);
} // namespace o2::its